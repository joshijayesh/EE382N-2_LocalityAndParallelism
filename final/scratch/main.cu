#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <chrono>
#include <fstream>

#include "commons.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

#define TOL pow(10, -12)


__global__
void identity(uint32_t n, uint32_t m, float* d) {
    uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if(y < n && x < m)
        d[y * m + x] = x == y ? 1 : 0;
}


inline void init_H(uint32_t n, float* H) {
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridDim((n + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (n + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

    identity<<<gridDim, blockDim>>> (n, n, H);
}


__global__
void vector_norm(uint32_t n, uint32_t full_n, float* R, float* V, float* T) {
    uint16_t wid = threadIdx.x >> THREADS_PER_WARP_LOG;
    uint16_t lane = threadIdx.x & THREADS_PER_WARP_MASK;
    uint32_t idx_x = threadIdx.x;
    float sum = 0.0;

    __shared__ float shared_sum[WARPS_PER_BLOCK];

    for(uint32_t i = idx_x; i < n; i += THREADS_PER_BLOCK) {
        float element = R[i * full_n];
        sum += element * element;
    }

    for(int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(FULL_WARP_MASK, sum, offset);
    }

    __syncthreads();

    if(lane == 0) {
        shared_sum[wid] = sum;
    }

    __syncthreads();

    if(wid == 0 && threadIdx.x < WARPS_PER_BLOCK) {
        sum = shared_sum[threadIdx.x];
        for(int offset = WARPS_PER_BLOCK / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xff, sum, offset);
        }

        if(threadIdx.x == 0) {
            sum = sqrt(sum);
            if((sum < 0 && R[0] > 0) || (sum > 0 && R[0] < 0))  // copy sign
                sum = -sum;
            shared_sum[0] = sum + R[0];
        }
    }

    __syncthreads();

    if(lane == 0) {
        sum = shared_sum[0];
    }

    sum = __shfl_sync(FULL_WARP_MASK, sum, 0);

    float tau = 0.0;

    for(uint32_t i = idx_x; i < n; i += THREADS_PER_BLOCK) {
        if(i != 0) {
            V[i] = R[i * full_n] / sum;

            tau += (V[i] * V[i]);
        } else {
            V[0] = 1;
            tau += 1;
        }
    }

    for(int offset = 16; offset > 0; offset /= 2) {
        tau += __shfl_down_sync(FULL_WARP_MASK, tau, offset);
    }

    __syncthreads();

    if(lane == 0) {
        shared_sum[wid] = tau;
    }

    __syncthreads();

    if(wid == 0 && threadIdx.x < WARPS_PER_BLOCK) {
        tau = shared_sum[threadIdx.x];
        for(int offset = WARPS_PER_BLOCK / 2; offset > 0; offset /= 2) {
            tau += __shfl_down_sync(0xff, tau, offset);
        }

        if(threadIdx.x == 0) {
            *T = 2.0 / tau;
        }
    }
}


inline void find_v(uint32_t n, uint32_t full_n, float* R, float* V, float* T) {
    dim3 blockDim(THREADS_PER_BLOCK);
    dim3 gridDim(1);

    vector_norm<<<gridDim, blockDim>>> (n, full_n, R, V, T);
    hipDeviceSynchronize();
}

__global__
void matmul(uint32_t n, uint32_t m, uint32_t p, float *A, float *B, float *C) {
    __shared__ float Ab[MATMUL_TILE_DIM][MATMUL_TILE_DIM + 1];
    __shared__ float Bb[MATMUL_TILE_DIM][MATMUL_TILE_DIM + 1];

    uint32_t idx_x = (blockIdx.x * MATMUL_TILE_DIM) + threadIdx.x;
    uint32_t idx_y = (blockIdx.y * MATMUL_TILE_DIM) + threadIdx.y;

    float C_temp[4] = {0.0, 0.0, 0.0, 0.0};

    for (uint32_t tile = 0; tile < m; tile += MATMUL_TILE_DIM) {
        for(uint32_t j = 0; j < MATMUL_TILE_DIM; j += MATMUL_BLOCK_DIM_Y) {
            if(tile + threadIdx.x < m && (idx_y + j) < n)
                Ab[threadIdx.y + j][threadIdx.x] = A[(idx_y + j) * m + tile + threadIdx.x];
            else
                Ab[threadIdx.y + j][threadIdx.x] = 0.0;

            if(tile + threadIdx.y + j < m && idx_x < p)
                Bb[threadIdx.y + j][threadIdx.x] = B[(tile + threadIdx.y + j) * p + idx_x];
            else
                Bb[threadIdx.y + j][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for(uint32_t j = 0; j < MATMUL_TILE_DIM; j += MATMUL_BLOCK_DIM_Y) {
            for(uint32_t i = 0; i < MATMUL_BLOCK_DIM_X; i += 1) {
                C_temp[j / MATMUL_BLOCK_DIM_Y] += Ab[threadIdx.y + j][i] * Bb[i][threadIdx.x];
            }
        }

        __syncthreads();
    }

    for(uint32_t j = 0; j < MATMUL_TILE_DIM; j += MATMUL_BLOCK_DIM_Y) {
        if((idx_y + j) < n && idx_x < p)  {
            C[(idx_y + j) * p + idx_x] = C_temp[j / MATMUL_BLOCK_DIM_Y];
        }
    }
}

__global__
void transpose_kernel(uint32_t n, uint32_t m, float *A, float *A_t) {
    __shared__ float block[TRANSPOSE_TILE][TRANSPOSE_TILE + 1];

    uint16_t idx_x = (blockIdx.x * TRANSPOSE_TILE) + threadIdx.x;
    uint16_t idx_y = (blockIdx.y * TRANSPOSE_TILE) + threadIdx.y;

    for(int j = 0; j < TRANSPOSE_TILE; j += TRANSPOSE_BLOCK_DIM_Y)
        if((idx_y + j) < m && idx_x < n)
            block[threadIdx.y + j][threadIdx.x] = A[(idx_y + j) * n + idx_x];

    __syncthreads();

    idx_x = (blockIdx.y * TRANSPOSE_TILE) + threadIdx.x;
    idx_y = (blockIdx.x * TRANSPOSE_TILE) + threadIdx.y;

    for(int j = 0; j < TRANSPOSE_TILE; j += TRANSPOSE_BLOCK_DIM_Y)
        if((idx_y + j) < n && idx_x < m)
            A_t[(idx_y + j) * m + idx_x] = block[threadIdx.x][threadIdx.y + j];
}

__global__
void matmul_vectors_herm(uint32_t full_n, uint32_t n, uint32_t j, float* A, float *C, float *T) {
    uint32_t idx_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint32_t idx_y = (blockIdx.y * blockDim.x) + threadIdx.y;

    if(idx_y < n && idx_x < n) {
        C[(idx_y + j) * full_n + j + idx_x] -= A[idx_y] * A[idx_x] * T[0];
    }
}

inline void find_herm(uint32_t n, uint32_t j, float* H, float* V, float* T) {
    uint32_t m = n - j;

    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridDim((m + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (m + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

    matmul_vectors_herm<<<gridDim, blockDim>>> (n, m, j, V, H, T);
    hipDeviceSynchronize();
}

inline void find_Q_R(uint32_t n, float* R, float* Q, float* H, float* R_o, float* Q_o) {
    dim3 blockDim(MATMUL_BLOCK_DIM_X, MATMUL_BLOCK_DIM_Y);
    dim3 gridDim((n + MATMUL_TILE_DIM - 1) / MATMUL_TILE_DIM, (n + MATMUL_TILE_DIM - 1) / MATMUL_TILE_DIM);

    matmul<<<gridDim, blockDim>>>(n, n, n, H, R, R_o);
    matmul<<<gridDim, blockDim>>>(n, n, n, H, Q, Q_o);
    hipDeviceSynchronize();
}

inline void transpose_Q(uint32_t n, float* Q, float* Q_o) {
    dim3 blockDim(TRANSPOSE_BLOCK_DIM_X, TRANSPOSE_BLOCK_DIM_Y);
    dim3 gridDim((n + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE, (n + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE);

    transpose_kernel<<<gridDim, blockDim>>> (n, n, Q_o, Q);
    hipDeviceSynchronize();
}


void qr_decomposition(uint32_t n, float* A, float* Q, float* R, float* H, float* V, float* T, float* R_o, float* Q_o) {
    hipMemcpy(R, A, sizeof(float) * n * n, hipMemcpyDeviceToDevice);
    init_H(n, Q);

    for(int j = 0; j < n; j += 1) {
        if(j % 2 == 0) {
            init_H(n, H);
            find_v(n - j, n, &R[j * n + j], V, T);
            find_herm(n, j, H, V, T);
            find_Q_R(n, R, Q, H, R_o, Q_o);
        } else {
            init_H(n, H);
            find_v(n - j, n, &R_o[j * n + j], V, T);
            find_herm(n, j, H, V, T);
            find_Q_R(n, R_o, Q_o, H, R, Q);
        }
    }

    if(n % 2 == 1) {
        hipMemcpy(R, R_o, sizeof(float) * n * n, hipMemcpyDeviceToDevice);
    } else {
        hipMemcpy(Q_o, Q, sizeof(float) * n * n, hipMemcpyDeviceToDevice);
    }

    transpose_Q(n, Q, Q_o);
}

inline void new_A(uint32_t n, float* R, float* Q, float* A) {
    dim3 blockDim(MATMUL_BLOCK_DIM_X, MATMUL_BLOCK_DIM_Y);
    dim3 gridDim((n + MATMUL_TILE_DIM - 1) / MATMUL_TILE_DIM, (n + MATMUL_TILE_DIM - 1) / MATMUL_TILE_DIM);

    matmul<<<gridDim, blockDim>>>(n, n, n, R, Q, A);
    hipDeviceSynchronize();
}

__global__
void vector_norm_diag(uint32_t n, float* A, float* A_i, float* T) {
    uint16_t wid = threadIdx.x >> THREADS_PER_WARP_LOG;
    uint16_t lane = threadIdx.x & THREADS_PER_WARP_MASK;
    uint32_t idx_x = threadIdx.x;
    float sum = 0.0;

    __shared__ float shared_sum[WARPS_PER_BLOCK];

    for(uint32_t i = idx_x; i < n; i += THREADS_PER_BLOCK) {
        float element = A[i * n + i] - A_i[i * n + i];
        sum += element * element;
    }

    for(int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(FULL_WARP_MASK, sum, offset);
    }

    __syncthreads();

    if(lane == 0) {
        shared_sum[wid] = sum;
    }

    __syncthreads();

    if(wid == 0 && threadIdx.x < WARPS_PER_BLOCK) {
        sum = shared_sum[threadIdx.x];
        for(int offset = WARPS_PER_BLOCK / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xff, sum, offset);
        }

        if(threadIdx.x == 0) {
            sum = sqrt(sum);
            *T = sum;
        }
    }
}

inline void find_ev_norm(uint32_t n, float* A, float* A_i, float* Norm) {
    dim3 blockDim(THREADS_PER_BLOCK);
    dim3 gridDim(1);

    vector_norm_diag<<<gridDim, blockDim>>>(n, A, A_i, Norm);
    hipDeviceSynchronize();
}

void qr_iteration(uint32_t n, float* A, float* Q) {
    float* d_R;
    float* d_H;
    float* d_A_i;
    float* d_V;
    float* d_T;
    float* d_R_o;
    float* d_Q_o;
    float* d_norm;
    hipMalloc((void**) &d_R, sizeof(float) * n * n);
    hipMalloc((void**) &d_A_i, sizeof(float) * n * n);
    hipMalloc((void**) &d_R_o, sizeof(float) * n * n);
    hipMalloc((void**) &d_Q_o, sizeof(float) * n * n);
    hipMalloc((void**) &d_H, sizeof(float) * n * n);
    hipMalloc((void**) &d_V, sizeof(float) * n);
    hipMalloc((void**) &d_T, sizeof(float));
    hipMalloc((void**) &d_norm, sizeof(float));

    float h_norm;

    for(int i = 0; i < n*n; i += 1) {
        hipMemcpyAsync(d_A_i, A, sizeof(float) * n * n, hipMemcpyDeviceToDevice);
        qr_decomposition(n, A, Q, d_R, d_H, d_V, d_T, d_R_o, d_Q_o);
        new_A(n, d_R, Q, A);
        find_ev_norm(n, A, d_A_i, d_norm);

        hipMemcpy(&h_norm, d_norm, sizeof(float), hipMemcpyDeviceToHost);

        if(h_norm < TOL) break;
    }

    hipFree(d_R);
    hipFree(d_H);
    hipFree(d_V);
    hipFree(d_T);
    hipFree(d_R_o);
    hipFree(d_Q_o);
    hipFree(d_A_i);
    hipFree(d_norm);
}

float* read_arr(uint32_t n, std::string file_name) {
    float* arr;
    float temp;

    arr = (float*) malloc(sizeof(float) * n * n);

    std::ifstream file(file_name);
    
    for(int i = 0; i < n * n; i += 1 ){
        file >> temp;
        arr[i] = temp;
    }

    file.close();

    return arr;
}


int main(int argc, char *argv[]) {
    float* d_A;
    float* d_Q;

    float* arr = read_arr(240, argv[1]);

    uint32_t n = 240;
    /*
    float arr[16] = {0.12857085, 0.45785891, 0.52853578, 0.14088372,
                      0.68806621, 0.50025996, 0.31960012, 0.09496113,
                        0.44442504, 0.36004189, 0.59893727, 0.32767798,
                         0.61017576, 0.4750934,  0.12999719, 0.90647875};
    */


    hipMalloc((void**) &d_A, sizeof(float) * n * n);
    hipMalloc((void**) &d_Q, sizeof(float) * n * n);

    hipMemcpy(d_A, arr, sizeof(float) * n * n, hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    qr_iteration(n, d_A, d_Q);
    auto stop = std::chrono::high_resolution_clock::now();

    std::cout << std::chrono::duration_cast<std::chrono::seconds>(stop - start).count() << " seconds" << std::endl;

    /*
    hipMemcpy(h_A, d_A, sizeof(float) * n * n, hipMemcpyDeviceToHost);

    std::cout << "A" << std::endl;
    for(int i = 0; i < n; i += 1) {
        for(int k = 0; k < n; k += 1) {
            std::cout << h_A[i * n + k] << " ";
        }
        std::cout << std::endl;
    }
    */

    hipFree(d_A);
    hipFree(d_Q);

    free(arr);
} 
