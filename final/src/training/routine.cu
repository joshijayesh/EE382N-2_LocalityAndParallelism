#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "commons.hpp"
#include "pgm/pgm.hpp"
#include "training/routine.hpp"

#include "training/routine.cuh"
#include "training/kernels.cuh"

#include "checker/routine_test.cuh"


void PCARoutine::load_matrix() {
    int device_count;

    CUDAERR_CHECK(
        hipGetDeviceCount(&device_count),
        "Unable to read CUDA Device Count", ERR_CUDA_GET_DEVICE);

    std::cout << "Num CUDA Devices: " << device_count << std::endl;

    // Allocate the matrix on the GPU
    CUDAERR_CHECK(
        hipMalloc((void **) &d_data, sizeof(float) * width * height * num_images),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    // Allocate space for mean image
    CUDAERR_CHECK(
        hipMalloc((void **) &d_mean, sizeof(float) * width * height),
        "Unable to malloc d_mean", ERR_CUDA_MALLOC);
    
    // Copy over data to the GPU
    int i = 0;
    for (PGMData img : pgm_list) {
        CUDAERR_CHECK(
            hipMemcpy(d_data + (i++ * width * height),
                       img.matrix,
                       sizeof(float) * width * height,
                       hipMemcpyHostToDevice),
            "Unable to copy matrices to device!", ERR_CUDA_MEMCPY);
    }

    // Allocate params
    DeviceConstants params;
    params.width = width;
    params.height = height;
    params.num_images = num_images;
    params.data = d_data;
    params.mean = d_mean;
    params.image_size = width * height;

    CUDAERR_CHECK(
        hipMemcpyToSymbol(HIP_SYMBOL(pca_dev_params), &params, sizeof(DeviceConstants)),
        "Unable to copy device constants to device!", ERR_CUDA_MEMCPY);

    std::cout << "Finished GPU vars" << std::endl;
}

void PCARoutine::mean_image() {
    // 1 warp per pixel
    uint32_t nx = (width + WARPS_PER_BLOCK) / WARPS_PER_BLOCK;
    dim3 blocks2D (nx, height);
    dim3 grid2D (THREADS_PER_BLOCK, 1);

    mean_reduce<<<blocks2D, grid2D>>> ();
    hipDeviceSynchronize();

    mean_checker(width, height, pgm_list, d_mean);
}

void PCARoutine::subtract() {

}

void PCARoutine::transpose() {

}

void PCARoutine::matmul() {

}

PCARoutine::~PCARoutine() {
    if (d_data) {
        std::cout << "Cleaning up~" << std::endl;
        hipFree(d_data);
        hipFree(d_mean);
    }
}

