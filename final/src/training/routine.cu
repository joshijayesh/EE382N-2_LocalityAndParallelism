#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "commons.hpp"
#include "pgm/pgm.hpp"
#include "training/routine.hpp"

#include "training/routine.cuh"

__constant__ DeviceConstants pca_dev_params;


void PCARoutine::load_matrix() {
    int device_count;

    CUDAERR_CHECK(
        hipGetDeviceCount(&device_count),
        "Unable to read CUDA Device Count", ERR_CUDA_GET_DEVICE);

    std::cout << "Num CUDA Devices: " << device_count << std::endl;

    hipMalloc(&d_data, sizeof(uint8_t) * width * height * num_images);

    int i = 0;
    for (PGMData img : pgm_list) {
        CUDAERR_CHECK(
            hipMemcpy(d_data + (i++ * width * height),
                       img.matrix,
                       sizeof(uint8_t) * width * height,
                       hipMemcpyHostToDevice),
            "Unable to copy matrices to device!", ERR_CUDA_MEMCPY);
    }

    DeviceConstants params;
    params.width = width;
    params.height = height;
    params.num_images = num_images;
    params.data = d_data;

    CUDAERR_CHECK(
        hipMemcpyToSymbol(HIP_SYMBOL(pca_dev_params), &params, sizeof(DeviceConstants)),
        "Unable to copy device constants to device!", ERR_CUDA_MEMCPY);
}

void PCARoutine::mean_image() {

}

void PCARoutine::subtract() {

}

void PCARoutine::transpose() {

}

void PCARoutine::matmul() {

}

PCARoutine::~PCARoutine() {
    if (d_data) {
        hipFree(d_data);
    }
}

