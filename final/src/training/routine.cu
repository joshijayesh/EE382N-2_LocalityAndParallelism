#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "commons.hpp"
#include "pgm/pgm.hpp"
#include "training/routine.hpp"

#include "training/routine.cuh"
#include "training/kernels.cuh"

#include "checker/routine_test.cuh"

// After some digging __constant__ is not scalable across multiple files... this thing sucks~
// Hence now just use const ptrs passed to each kernel
// __constant__ DeviceConstants pca_dev_params;


void PCARoutine::load_matrix() {
    int device_count;

    CUDAERR_CHECK(
        hipGetDeviceCount(&device_count),
        "Unable to read CUDA Device Count", ERR_CUDA_GET_DEVICE);

    std::cout << "Num CUDA Devices: " << device_count << std::endl;

    // Allocate the matrix on the GPU
    CUDAERR_CHECK(
        hipMalloc((void **) &d_data, sizeof(float) * width * height * num_images),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);


    CUDAERR_CHECK(
        hipMalloc((void **) &d_params, sizeof(DeviceConstants)),
        "Unable to malloc d_params", ERR_CUDA_MALLOC);
    
    // Copy over data to the GPU
    int i = 0;
    for (PGMData img : pgm_list) {
        CUDAERR_CHECK(
            hipMemcpy(d_data + (i++ * width * height),
                       img.matrix,
                       sizeof(float) * width * height,
                       hipMemcpyHostToDevice),
            "Unable to copy matrices to device!", ERR_CUDA_MEMCPY);
    }

    // Allocate params
    DeviceConstants params;
    params.width = width;
    params.height = height;
    params.num_images = num_images;
    params.data = d_data;
    params.image_size = width * height;

    CUDAERR_CHECK(
        hipMemcpy(d_params,
                   &params,
                   sizeof(DeviceConstants),
                   hipMemcpyHostToDevice),
        "Unable to copy device constants to device!", ERR_CUDA_MEMCPY);

    std::cout << "Finished GPU vars" << std::endl;
}

// Calculates mean and subtracts from each image yielding A
void PCARoutine::mean_image() {
    // 1 warp per pixel
    uint32_t nx = (width + WARPS_PER_BLOCK) / WARPS_PER_BLOCK;
    dim3 blocks2D (nx, height);
    dim3 grid2D (THREADS_PER_BLOCK, 1);

    mean_reduce<<<blocks2D, grid2D>>> ((DeviceConstants *)(d_params));
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    mean_checker(width, height, pgm_list, d_data);
    #endif
}

void PCARoutine::transpose() {

}

void PCARoutine::matmul() {

}

PCARoutine::~PCARoutine() {
    if (d_data) {
        std::cout << "Cleaning up~" << std::endl;
        hipFree(d_data);
        hipFree(d_params);
    }
}

