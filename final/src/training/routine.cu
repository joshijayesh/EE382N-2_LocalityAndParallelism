#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "commons.hpp"
#include "pgm/pgm.hpp"
#include "training/routine.hpp"

#include "training/routine.cuh"
#include "training/kernels.cuh"

#include "checker/routine_test.cuh"

// After some digging __constant__ is not scalable across multiple files... this thing sucks~
// Hence now just use const ptrs passed to each kernel
// __constant__ DeviceConstants pca_dev_params;


void PCARoutine::load_matrix() {
    int device_count;

    CUDAERR_CHECK(
        hipGetDeviceCount(&device_count),
        "Unable to read CUDA Device Count", ERR_CUDA_GET_DEVICE);

    std::cout << "Num CUDA Devices: " << device_count << std::endl;

    // Allocate the matrix on the GPU
    CUDAERR_CHECK(
        hipMalloc((void **) &d_data, sizeof(float) * width * height * num_images),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_data_temp, sizeof(float) * width * height * num_images),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_data_transpose, sizeof(float) * width * height * num_images),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    // cov = (width * num images)^2 -- This is hugeee!
    CUDAERR_CHECK(
        hipMalloc((void **) &d_data_cov, sizeof(float) * (num_images * num_images)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_eigenvectors, sizeof(float) * (num_images * num_images)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);
   
     CUDAERR_CHECK(
        hipMalloc((void **) &d_eigenvalues, sizeof(float) * (num_images * num_images)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_eigenvectors_sorted, sizeof(float) * (num_images * num_images)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_real_eigenvectors, sizeof(float) * (width * height * num_components)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_real_eigenvectors_norm, sizeof(float) * (width * height * num_components)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_real_eigenvectors_transpose, sizeof(float) * (width * height * num_components)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

    CUDAERR_CHECK(
        hipMalloc((void **) &d_results, sizeof(float) * (num_images * num_components)),
        "Unable to malloc d_data", ERR_CUDA_MALLOC);

   CUDAERR_CHECK(
        hipMalloc((void **) &d_params, sizeof(DeviceConstants)),
        "Unable to malloc d_params", ERR_CUDA_MALLOC);
    
    // Copy over data to the GPU
    int i = 0;
    for (PGMData img : pgm_list) {
        CUDAERR_CHECK(
            hipMemcpy(d_data_temp + (i++ * width * height),
                       img.matrix,
                       sizeof(float) * width * height,
                       hipMemcpyHostToDevice),
            "Unable to copy matrices to device!", ERR_CUDA_MEMCPY);
    }

    // Allocate params
    DeviceConstants params;
    params.width = width;
    params.height = height;
    params.m = height * width;
    params.n = num_images;
    params.num_images = num_images;
    params.data = d_data_temp;
    params.A = d_data;
    params.A_t = d_data_transpose;
    params.image_size = width * height;

    CUDAERR_CHECK(
        hipMemcpy(d_params,
                   &params,
                   sizeof(DeviceConstants),
                   hipMemcpyHostToDevice),
        "Unable to copy device constants to device!", ERR_CUDA_MEMCPY);

    std::cout << "Finished GPU vars" << std::endl;
    std::cout << "width = " << width << std::endl;
    std::cout << "height = " << height << std::endl;
    std::cout << "num_images = " << num_images << std::endl;
    std::cout << "n = " << params.n << std::endl;
    std::cout << "m = " << params.m << std::endl;
}

// Calculates mean and subtracts from each image yielding A
void PCARoutine::mean_image() {
    // 1 warp per pixel
    uint32_t nx = (width + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    dim3 blocks2D (nx, height);
    dim3 grid2D (THREADS_PER_BLOCK, 1);

    mean_reduce<<<blocks2D, grid2D>>> (width, width * height, num_images, d_data_temp, d_data);
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    mean_checker(width, height, pgm_list, d_data);
    #endif
}

void PCARoutine::compute_covariance() {
    uint32_t n = num_images;
    uint32_t m = width * height;

    dim3 block2D (((n + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE), ((m + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE));
    dim3 grid2D (TRANSPOSE_BLOCK_DIM_X, TRANSPOSE_BLOCK_DIM_Y);

    transpose_kernel<<<block2D, grid2D>>> (n, m, d_data, d_data_transpose);
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    transpose_checker(n, m, d_data, d_data_transpose);
    #endif

    dim3 m_block2D (((n + MATMUL_TILE_DIM - 1) / MATMUL_TILE_DIM), ((n + MATMUL_TILE_DIM - 1) / MATMUL_TILE_DIM));
    dim3 m_grid2D (MATMUL_BLOCK_DIM_X, MATMUL_BLOCK_DIM_Y);

    matmul<<<m_block2D, m_grid2D>>> (n, m, n, d_data_transpose, d_data, d_data_cov);
    identity_matrix<<<m_block2D, m_grid2D>>> (n, d_eigenvectors);
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    matmul_checker(n, m, n, d_data_transpose, d_data, d_data_cov);
    #endif
}

void PCARoutine::sort_eigenvectors() {
    uint32_t n = num_images;
    float* v = d_eigenvectors;
    float* w = d_eigenvalues;

    int* sort_index,*sort_index_copy;

    float *w_1d;
    float *w_1d_copy;
    float *v_sorted = d_eigenvectors_sorted;

    hipMalloc((void **) &w_1d, sizeof(float)*n);
    hipMalloc((void **) &w_1d_copy, sizeof(float)*n);

    hipMalloc((void **) &sort_index, sizeof(int) * n);
    hipMalloc((void **) &sort_index_copy, sizeof(int) * n);

    dim3 blockDim(256,1);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

    sort_initialize<<<gridDim,blockDim>>>(n,sort_index,w_1d,w);
    hipDeviceSynchronize();

    sort_value_kernel<<<1,1>>>(w_1d,w_1d_copy,sort_index,sort_index_copy,n);
    hipDeviceSynchronize();

    sort_vector_kernel<<<gridDim,blockDim>>>(v,v_sorted,sort_index,n);
    hipDeviceSynchronize();

    hipFree(sort_index);
    hipFree(sort_index_copy);
    hipFree(w_1d);
    hipFree(w_1d_copy);

    return;
}


void PCARoutine::post_process() {
    uint32_t n = num_images;
    uint32_t m = width * height;
    uint32_t p = num_components;

    dim3 block2D (((p + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE), ((m + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE));
    dim3 grid2D (MATMUL_BLOCK_DIM_X, MATMUL_BLOCK_DIM_Y);

    // U = A * V
    matmul<<<block2D, grid2D>>> (m, n, p, d_data, d_eigenvectors_sorted, d_real_eigenvectors);
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    matmul_checker(m, n, p, d_data, d_eigenvectors_sorted, d_real_eigenvectors);
    #endif

    dim3 block2D_3 (m, p);
    dim3 grid2D_3 (THREADS_PER_BLOCK, 1);

    // Normalize squared sum
    norm_squaredsum<<<block2D_3, grid2D_3>>> (m, p, d_real_eigenvectors, d_real_eigenvectors_norm);
    hipDeviceSynchronize();

    // Transpose for projection
    transpose_kernel<<<block2D, grid2D>>> (p, m, d_real_eigenvectors_norm, d_real_eigenvectors_transpose);
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    transpose_checker(p, m, d_real_eigenvectors_norm, d_real_eigenvectors_transpose);
    #endif

    dim3 block2D_2 (((n + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE), ((p + TRANSPOSE_TILE - 1) / TRANSPOSE_TILE));
    dim3 grid2D_2 (TRANSPOSE_BLOCK_DIM_X, TRANSPOSE_BLOCK_DIM_Y);

    // Projection: Gamma = U_T * A
    matmul<<<block2D_2, grid2D_2>>> (p, m, n, d_real_eigenvectors_transpose, d_data, d_results);
    hipDeviceSynchronize();

    #ifdef EN_CHECKER
    matmul_checker(p, m, n, d_real_eigenvectors_transpose, d_data, d_results);
    #endif
}

PCARoutine::~PCARoutine() {
    if (d_data) {
        std::cout << "Cleaning up~" << std::endl;
        hipFree(d_data);
        hipFree(d_data_temp);
        hipFree(d_data_transpose);
        hipFree(d_data_cov);
        hipFree(d_eigenvectors);
        hipFree(d_eigenvectors_sorted);
	    hipFree(d_eigenvalues);
	    hipFree(d_real_eigenvectors);
	    hipFree(d_real_eigenvectors_norm);
        hipFree(d_real_eigenvectors_transpose);
        hipFree(d_params);
        hipFree(d_results);
    }
}

