#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "training/routine.cuh"


__global__
void matmul(uint32_t n, uint32_t m, uint32_t p, float *A, float *B, float *C) {
    __shared__ float Ab[MATMUL_TILE_DIM][MATMUL_TILE_DIM + 1];
    __shared__ float Bb[MATMUL_TILE_DIM][MATMUL_TILE_DIM + 1];

    uint16_t idx_x = (blockIdx.x * MATMUL_TILE_DIM) + threadIdx.x;
    uint16_t idx_y = (blockIdx.y * MATMUL_TILE_DIM) + threadIdx.y;


    float C_temp[4] = {0.0, 0.0, 0.0, 0.0};

    for (uint16_t tile = 0; tile < m; tile += MATMUL_TILE_DIM) {
        for(int j = 0; j < MATMUL_TILE_DIM; j += MATMUL_BLOCK_DIM_Y) {
            if(tile + idx_x < m && (idx_y + j) < n)
                Ab[threadIdx.y + j][threadIdx.x] = A[(idx_y + j) * m + tile + idx_x];
            else
                Ab[threadIdx.y + j][threadIdx.x] = 0.0;

            if(tile + idx_x < m && (idx_y + j) < p)
                Bb[threadIdx.y + j][threadIdx.x] = B[(tile + idx_x) * p + (idx_y + j)];
            else
                Bb[threadIdx.y + j][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for(uint16_t j = 0; j < MATMUL_TILE_DIM; j += MATMUL_BLOCK_DIM_Y) {
            for(uint16_t i = 0; i < MATMUL_BLOCK_DIM_X; i += 1) {
                C_temp[j / MATMUL_BLOCK_DIM_Y] += Ab[threadIdx.y + j][i] * Bb[i][threadIdx.x];
            }
        }

        __syncthreads();
    }

    for(uint16_t j = 0; j < MATMUL_TILE_DIM; j += MATMUL_BLOCK_DIM_Y) {
        if((idx_y + j) < n && idx_x < p)  {
            C[(idx_y + j) * p + idx_x] = C_temp[j / MATMUL_BLOCK_DIM_Y];
        }
    }
}

