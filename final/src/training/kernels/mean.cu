#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "training/routine.cuh"


__global__
void mean_reduce(const DeviceConstants *pca_dev_params) {
    uint16_t wid = threadIdx.x >> THREADS_PER_WARP_LOG;
    uint16_t lane = threadIdx.x & THREADS_PER_WARP_MASK;
    uint32_t idx_x = ((blockIdx.x * WARPS_PER_BLOCK) + wid);

    // 1 warp per pixel
    if(idx_x < pca_dev_params->width) {
        uint32_t pixel = (blockIdx.y * pca_dev_params->width) + idx_x;
        uint32_t stride = pca_dev_params->image_size;
        float *img_ptr = pca_dev_params->data + pixel + (lane * stride);

        uint16_t img_num = lane;
        float sum = 0.0;

        
        // Noted this is a bit inefficient if num_images < 32
        // But aint noone runing actual training with that little training set
        while(img_num < pca_dev_params->num_images) {
            sum += *img_ptr;
            img_ptr += (stride * THREADS_PER_WARP);
            img_num += THREADS_PER_WARP;
        }
        
        for(int offset = 16; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(FULL_WARP_MASK, sum, offset);
        }

        if(lane == 0) {
            sum = sum / pca_dev_params->num_images;
        }

        float mean = __shfl_sync(FULL_WARP_MASK, sum, 0);

        img_num = lane;
        img_ptr = pca_dev_params->data + pixel + (lane * stride);

        uint32_t A_pixel = ((blockIdx.y * pca_dev_params->width) + idx_x) * pca_dev_params->num_images;
        uint32_t A_stride = 1;
        float *A_ptr = pca_dev_params->A + A_pixel + (lane * A_stride);

        while(img_num < pca_dev_params->num_images) {
            *A_ptr = *img_ptr - mean;
            img_ptr += (stride * THREADS_PER_WARP);
            A_ptr += (A_stride * THREADS_PER_WARP);
            img_num += THREADS_PER_WARP;
        }
    }
}

__global__
void norm_squaredsum(uint32_t n, uint32_t m, float *in, float *out) {
    float sum = 0.0;
    uint32_t vector_num = blockIdx.y;

    uint16_t wid = threadIdx.x >> THREADS_PER_WARP_LOG;
    uint16_t lane = threadIdx.x & THREADS_PER_WARP_MASK;

    __shared__ float shared_sum[WARPS_PER_BLOCK];

    for(uint32_t offset = threadIdx.x; offset < m; offset += THREADS_PER_BLOCK) {
        uint32_t element = offset * m + vector_num;
        sum += in[element] * in[element];
    }

    for(int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(FULL_WARP_MASK, sum, offset);
    }

    __syncthreads();

    if(lane == 0) {
        shared_sum[wid] = sum;
    }

    __syncthreads();

    if(wid == 0 && threadIdx.x < WARPS_PER_BLOCK) {
        sum = shared_sum[threadIdx.x];
        for(int offset = WARPS_PER_BLOCK / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(FULL_WARP_MASK, sum, offset);
        }

        if(threadIdx.x == 0) {
            shared_sum[0] = sqrt(sum);
        }
    }

    __syncthreads();

    if(lane == 0) {
        sum = shared_sum[0];
    }

    sum = __shfl_sync(FULL_WARP_MASK, sum, 0);
    for(uint32_t offset = threadIdx.x; offset < m; offset += THREADS_PER_BLOCK) {
        uint32_t element = offset * m + vector_num;
        out[element] = in[element] / sum;
    }
}

