#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "commons.hpp"
#include "pgm/pgm.hpp"
#include "training/routine.hpp"

#include "training/routine.cuh"
#include "training/kernels.cuh"


void mean_checker(int width, int height, std::vector<PGMData> pgm_list, float* A) {
    float *data;
    bool fail = false;

    data = (float *) malloc(sizeof(float) * (width * height) * pgm_list.size());

    CUDAERR_CHECK(
        hipMemcpy(data,
                   A,
                   sizeof(float) * width * height * pgm_list.size(),
                   hipMemcpyDeviceToHost),
        "Unable to copy data from device!", ERR_CUDA_MEMCPY);

    for (int i = 0; i < height; i += 1) {
        for(int j = 0; j < width; j += 1) {
            int pixel = (i * width) + j;
            float sum = 0.0;
            for(PGMData img : pgm_list) {
                sum += img.matrix[pixel];
            }
            sum /= pgm_list.size();

            int k = 0;
            int row = pixel * pgm_list.size();
            for(PGMData img : pgm_list) {
                float result = data[row + k++];
                if(result != img.matrix[pixel] - sum) {
                    std::cout << "Mean compare failed px " << pixel << "! Expected " << img.matrix[pixel] - sum << " Actual " << result << std::endl;
                    fail = true;
                }
            }
        }
    }

    CERR_CHECK(!fail, "Mean checker failed!!", ERR_CHECKER_FAILED);
    std::cout << "Mean checker passed!" << std::endl;

    free(data);
}


void transpose_checker(int width, int height, float* A, float* A_T) {
    float *data;
    float *data_T;
    bool fail = false;
    
    data = (float *) malloc(sizeof(float) * (width * height));
    data_T = (float *) malloc(sizeof(float) * (width * height)); 


    CUDAERR_CHECK(
        hipMemcpy(data,
                   A,
                   sizeof(float) * width * height,
                   hipMemcpyDeviceToHost),
        "Unable to copy data from device!", ERR_CUDA_MEMCPY);

    CUDAERR_CHECK(
        hipMemcpy(data_T,
                   A_T,
                   sizeof(float) * width * height,
                   hipMemcpyDeviceToHost),
        "Unable to copy data from device!", ERR_CUDA_MEMCPY);

    for(int n = 0; n < width * height; n += 1) {
        int i = n / height;
        int j = n % height;
        if(data_T[n] != data[width * j + i]) {
            std::cout << "Transpose failed i=" << i << ",j=" << j << "! Expected " << data[width * j + i] << " Actual " << data_T[n] << std::endl;
            fail = true;
        }
    }

    CERR_CHECK(!fail, "Transpose checker failed!!", ERR_CHECKER_FAILED);
    std::cout << "Transpose checker passed!" << std::endl;

    free(data);
    free(data_T);
}

