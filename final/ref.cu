#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include gesvdj_example.cpp 
 *   g++ -o gesvdj_example gesvdj_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <fstream>
#include <sys/time.h>
void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %20.16E\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    const int m = 240;
    const int n = 240;
    const int lda = m; /* A is m-by-n */
    const int ldu = m; /* U is m-by-m */
    const int ldv = n; /* V is n-by-n */
    const int minmn = min(m,n);
/*       | 1 2  |
 *   A = | 4 5  |
 *       | 2 1  |
 */
    double A[lda*n]; //= { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0};


    std::ifstream cov_file("dump_identity.txt");

   // char toss;

//    cov_file.read(&toss, 1);

    for(uint32_t i = 0; i < lda; i += 1) {
        for(uint32_t j = 0; j < n; j += 1) {
            cov_file >> A[i * n + j];
        }
    }

   std::ofstream file("dump_A.txt");
    for(int i = 0; i < lda; i += 1) {
        for(int j = 0; j < n; j += 1) {
            file << A[i * m + j] << " ";
        }
        file << std::endl;
    }


    // for(int i=0;i<lda;i++)
    //     for(int j=0;j<n;j++)
    //     A[i*n+j] = i+j-1;
    double U[ldu*m]; /* m-by-m unitary matrix, left singular vectors  */
    double V[ldv*n]; /* n-by-n unitary matrix, right singular vectors */
    double S[minmn];     /* numerical singular value */
/* exact singular values */
    double S_exact[minmn] = {7.065283497082729, 1.040081297712078};
    double *d_A = NULL;  /* device copy of A */
    double *d_S = NULL;  /* singular values */
    double *d_U = NULL;  /* left singular vectors */
    double *d_V = NULL;  /* right singular vectors */
    int *d_info = NULL;  /* error info */
    int lwork = 0;       /* size of workspace */
    double *d_work = NULL; /* devie workspace for gesvdj */
    int info = 0;        /* host copy of error info */


 
/* configuration of gesvdj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const int econ = 0 ; /* econ = 1 for economy size */

/* numerical results of gesvdj  */
    double residual = 0;
    int executed_sweeps = 0;

    printf("example of gesvdj \n");
    printf("tol = %E, default value is machine zero \n", tol);
    printf("max. sweeps = %d, default value is 100\n", max_sweeps);
    printf("econ = %d \n", econ);

    printf("A = (matlab base-1)\n");
    // printMatrix(m, n, A, lda, "A");
    printf("=====\n");

/* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: configuration of gesvdj */
    status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of tolerance is machine zero */
    status = hipsolverDnXgesvdjSetTolerance(
        gesvdj_params,
        tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of max. sweeps is 100 */
    status = hipsolverDnXgesvdjSetMaxSweeps(
        gesvdj_params,
        max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 3: copy A and B to device */
    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(double)*lda*n);
    cudaStat2 = hipMalloc ((void**)&d_S   , sizeof(double)*minmn);
    cudaStat3 = hipMalloc ((void**)&d_U   , sizeof(double)*ldu*m);
    cudaStat4 = hipMalloc ((void**)&d_V   , sizeof(double)*ldv*n);
    cudaStat5 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

 
/* step 4: query workspace of SVD */
    status = hipsolverDnDgesvdj_bufferSize(
        cusolverH,
        jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
              /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ, /* econ = 1 for economy size */
        m,    /* nubmer of rows of A, 0 <= m */
        n,    /* number of columns of A, 0 <= n  */
        d_A,  /* m-by-n */
        lda,  /* leading dimension of A */
        d_S,  /* min(m,n) */
              /* the singular values in descending order */
        d_U,  /* m-by-m if econ = 0 */
              /* m-by-min(m,n) if econ = 1 */
        ldu,  /* leading dimension of U, ldu >= max(1,m) */
        d_V,  /* n-by-n if econ = 0  */
              /* n-by-min(m,n) if econ = 1  */
        ldv,  /* leading dimension of V, ldv >= max(1,n) */
        &lwork,
        gesvdj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work , sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

/* step 5: compute SVD */
    struct timeval t1, t2;

gettimeofday(&t1, 0);
    status = hipsolverDnDgesvdj(
        cusolverH,
        jobz,  /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
               /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ,  /* econ = 1 for economy size */
        m,     /* nubmer of rows of A, 0 <= m */
        n,     /* number of columns of A, 0 <= n  */
        d_A,   /* m-by-n */
        lda,   /* leading dimension of A */
        d_S,   /* min(m,n)  */
               /* the singular values in descending order */
        d_U,   /* m-by-m if econ = 0 */
               /* m-by-min(m,n) if econ = 1 */
        ldu,   /* leading dimension of U, ldu >= max(1,m) */
        d_V,   /* n-by-n if econ = 0  */
               /* n-by-min(m,n) if econ = 1  */
        ldv,   /* leading dimension of V, ldv >= max(1,n) */
        d_work,
        lwork,
        d_info,
        gesvdj_params);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    gettimeofday(&t2, 0);

double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

printf("Time to generate:  %3.1f ms \n", time);

    cudaStat1 = hipMemcpy(U, d_U, sizeof(double)*ldu*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_V, sizeof(double)*ldv*n, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(S, d_S, sizeof(double)*minmn, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat5 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);

    if ( 0 == info ){
        printf("gesvdj converges \n");
    }else if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }else{
        printf("WARNING: info = %d : gesvdj does not converge \n", info );
    }

    printf("S = singular values (matlab base-1)\n");
    std::ofstream file1("dump_S.txt");
    for(int i = 0; i < minmn; i += 1) {
            file1 << S[i] << " ";

        file1 << std::endl;
    }
    // printMatrix(minmn, 1, S, minmn, "S");
    printf("=====\n");

    printf("U = left singular vectors (matlab base-1)\n");
    std::ofstream file2("dump_U.txt");
    for(int i = 0; i < ldu; i += 1) {
        for(int j = 0; j < m; j += 1) {
            file2 << U[i * m + j] << " ";
        }
        file2 << std::endl;
    }
    // printMatrix(m, m, U, ldu, "U");
    printf("=====\n");

    printf("V = right singular vectors (matlab base-1)\n");
    std::ofstream file3("dump_V.txt");
    for(int i = 0; i < ldv; i += 1) {
        for(int j = 0; j < n; j += 1) {
            file3 << V[i * n + j] << " ";
        }
        file3 << std::endl;
    }
    // printMatrix(n, n, V, ldv, "V");
    printf("=====\n");

/* step 6: measure error of singular value */
    double ds_sup = 0;
    for(int j = 0; j < minmn; j++){
        double err = fabs( S[j] - S_exact[j] );
        ds_sup = (ds_sup > err)? ds_sup : err;
    }
    printf("|S - S_exact|_sup = %E \n", ds_sup);

    status = hipsolverDnXgesvdjGetSweeps(
        cusolverH,
        gesvdj_params,
        &executed_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status = hipsolverDnXgesvdjGetResidual(
        cusolverH,
        gesvdj_params,
        &residual);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    printf("residual |A - U*S*V**H|_F = %E \n", residual );
    printf("number of executed sweeps = %d \n", executed_sweeps );

/*  free resources  */
    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    if (d_V    ) hipFree(d_V);
    if (d_info) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);

    hipDeviceReset();
    return 0;
}
