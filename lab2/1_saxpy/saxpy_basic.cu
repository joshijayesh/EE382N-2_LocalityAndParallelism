#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"
#include "common_cuda.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from dev_offsetition of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
  // TODO: implement and use this interface if necessary  
}

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
    // TODO: implement and use this interface if necessary  
    *xarray      = (float *) CHECK_HOST_MALLOC(malloc(size * sizeof(float)));
    *yarray      = (float *) CHECK_HOST_MALLOC(malloc(size * sizeof(float)));
    *resultarray = (float *) CHECK_HOST_MALLOC(malloc(size * sizeof(float)));
}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
  // TODO: implement and use this interface if necessary  
    free(xarray);
    free(yarray);
    free(resultarray);
}

void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary

    float *device_x;
    float *device_y;
    float *device_result;
    //
    // TODO: allocate device memory buffers on the GPU using
    // hipMalloc.  The started code issues warnings on build because
    // these buffers are used in the call to saxpy_kernel below
    // without being initialized.
    //
    CHECK_CUDA_ERROR(hipMalloc((void **) &device_x, total_elems * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **) &device_y, total_elems * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **) &device_result, total_elems * sizeof(float)));

    //
    // TODO: Compute number of thread blocks.
    // 
    uint32_t num_blocks = (total_elems + (threadsPerBlock - 1)) / threadsPerBlock;


    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO: copy input arrays to the GPU using hipMemcpy
    //
    CHECK_CUDA_ERROR(hipMemcpy(device_x, xarray, total_elems * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(device_y, yarray, total_elems * sizeof(float), hipMemcpyHostToDevice));
    double endH2DTime = CycleTimer::currentSeconds();
    double timeH2D = endH2DTime - startTime;


    //
    // TODO: insert time here to begin timing only the kernel
    //
    double startGPUTime = CycleTimer::currentSeconds();

    // run saxpy_kernel on the GPU
    saxpy_kernel <<<num_blocks, threadsPerBlock>>>(total_elems, alpha, device_x, device_y, device_result);

    //
    // TODO: insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();

    double endGPUTime = CycleTimer::currentSeconds();
    double timeKernel = endGPUTime - startGPUTime;
    
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    
    //
    // TODO: copy result from GPU using hipMemcpy
    //
    double startD2H = CycleTimer::currentSeconds();
    CHECK_CUDA_ERROR(hipMemcpyAsync(resultarray, device_result, total_elems * sizeof(float), hipMemcpyDeviceToHost));

    double endD2H = CycleTimer::currentSeconds();
    double timeD2H = endD2H - startD2H;

    

    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    totalTimeAvg   += overallDuration;
    timeKernelAvg  += timeKernel;
    timeCopyH2DAvg += timeH2D;
    timeCopyD2HAvg += timeD2H;

    //
    // TODO free memory buffers on the GPU
    //
    CHECK_CUDA_ERROR(hipFree(device_x));
    CHECK_CUDA_ERROR(hipFree(device_y));
    CHECK_CUDA_ERROR(hipFree(device_result));
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
